#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <iomanip>

#include "matmult_cpp.cuh"

__global__ void MatIsZero_kernel(Matrix A, int* flg)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < A.getHeight() && c < A.getWidth()) {
        atomicAnd(flg, A(r, c) == 0);
    }
}

bool Matrix::isZero() const
{
    if (this->elements_cudaMalloc) {
        auto [gridDim, blockDim] = getGridAndBlockDim();

        int* flg = nullptr;
        hipMallocManaged(&flg, sizeof(int));
        *flg = 1;
        MatIsZero_kernel<<<gridDim, blockDim>>>(*this, flg);
        hipDeviceSynchronize();
        bool result = *flg;
        hipFree(flg);
        return result;
    } else {
        for (int r = 0; r < height; r++) {
            for (int c = 0; c < width; c++) {
                if ((*this)(r, c) != 0) {
                    return false;
                }
            }
        }
        return true;
    }
}

// Matrix multiplication kernel called by Mat::multGPU() - basic version
__global__ void MatMult_cpp_naive(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    C(r, c, A.multElement(B, r, c));
}

// Matrix multiplication kernel called by Mat::multGPU() - optimized version
// Should be run this way:
// size_t sharedMemSize = 2 * BLOCK_SIZE * BLOCK_SIZE * sizeof(float); // Total size for As and Bs
// MatMult_optimized<<<gridDim, blockDim, sharedMemSize>>>(A, B, C);
__global__ void MatMult_cpp_optimized(Matrix MatA, Matrix MatB, Matrix MatC)
{
    extern __shared__ char sharedMemory[];

    int R = blockIdx.y;
    int C = blockIdx.x;
    int r = threadIdx.y;
    int c = threadIdx.x;
    int w = blockDim.x;
    int W = MatA.getWidth() / w;

    Matrix Asub(w, w, false);
    Matrix Bsub(w, w, false);
    Matrix Csub(w, w, false);

    Matrix Asub_s(w, w, (float*)sharedMemory);
    Matrix Bsub_s(w, w, (float*)&(sharedMemory[w * w * sizeof(float)]));

    MatC.getSubMatrix(R, C, w, Csub);
    // Each thread computes one element of Csub
    float Csub_rc = 0;
    // C_{R,C} = \sum_{K=0}^{W-1} A_{R,K} B_{K,C}
    for (int K = 0; K < W; K++) {
        MatA.getSubMatrix(R, K, w, Asub);
        MatB.getSubMatrix(K, C, w, Bsub);
        __syncthreads();
        Asub.copyElement(Asub_s, r, c);
        Bsub.copyElement(Bsub_s, r, c);
        __syncthreads();
        // Csub_{r,c} = \sum_{k=0}^{w-1} A_{r,k} B_{k,c}
        Csub_rc += Asub_s.multElement(Bsub_s, r, c);
    }
    Csub(r, c, Csub_rc);
}

void Matrix::multGPU(const Matrix& A, const Matrix& B, bool optimized)
{
    assert(A.width == B.height);
    assert(A.height == this->height);
    assert(B.width == this->width);

    // Load A to device memory
    Matrix d_A(A.height, A.width, false, true);
    hipMemcpy(d_A.elements, A.elements, A.sizeInBytes(), hipMemcpyHostToDevice);

    // Load B to device memory
    Matrix d_B(B.height, B.width, false, true);
    hipMemcpy(d_B.elements, B.elements, B.sizeInBytes(), hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C(this->height, this->width, false, true);

    auto [dimGrid, dimBlock] = this->getGridAndBlockDim();

#ifndef NDEBUG
    assert(d_C.isZero());
#endif

    if (optimized) {
        size_t sharedMemSize = 2 * dimBlock.x * dimBlock.y * sizeof(float); // Total size for As and Bs
        MatMult_cpp_optimized<<<dimGrid, dimBlock, sharedMemSize>>>(d_A, d_B, d_C);
    } else {
        MatMult_cpp_naive<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    }

#ifndef NDEBUG
    bool isAZero = d_A.isZero();
    bool isBZero = d_B.isZero();
    bool isCZero = d_C.isZero();
    if (isAZero || isBZero) {
        assert(isCZero);
    } else {
        assert(!isCZero);
    }
#endif

    // Read C from device memory
    hipMemcpy(this->elements, d_C.elements, this->sizeInBytes(), hipMemcpyDeviceToHost);

#ifndef NDEBUG
    assert(isZero() == isCZero);
#endif
}

void Matrix::multHost(const Matrix& A, const Matrix& B)
{
    assert(A.width == B.height);
    assert(A.height == this->height);
    assert(B.width == this->width);

    for (int r = 0; r < A.height; r++) {
        for (int c = 0; c < B.width; c++) {
            (*this)(r, c, A.multElement(B, r, c));
        }
    }
}

std::ostream& operator<<(std::ostream& os, const Matrix& A)
{
    // Save the current format settings
    std::ios oldState(nullptr);
    oldState.copyfmt(os);

    os << A.name << " = [\n";
    for (int i = 0; i < A.height; ++i) {
        for (int j = 0; j < A.width; ++j) {
            os << std::fixed << std::setw(5) << std::setprecision(1) << A(i, j);
            if (j < A.width - 1) {
                os << " ";
            }
        }
        os << "\n";
    }
    os << "]\n";

    // Restore the original format settings
    os.copyfmt(oldState);
    return os;
}

bool Matrix::equal(const Matrix& B, float tol) const
{
    if (height != B.height || width != B.width)
        return false;
    for (int r = 0; r < height; r++)
        for (int c = 0; c < width; c++) {
            if (fabs((*this)(r, c) - B(r, c)) > tol)
                return false;
        }
    return true;
}
